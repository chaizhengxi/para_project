#include "csr.hpp"
#include "cuda_kernel.cuh"
#include <vector>
#include <iostream>

int main(int argc, char** argv) {
    size_t n = (argc > 1) ? std::stoul(argv[1]) : 1<<14;
    CSR hA = make_random_csr(n);
    std::vector<float> hx(n, 1.f), hy(n, 0.f);

    int nnz = hA.val.size();
    // device memory
    int  *d_rowptr, *d_colind;
    float *d_val, *d_x, *d_y;
    hipMalloc(&d_rowptr,(n+1)*sizeof(int));
    hipMalloc(&d_colind,nnz*sizeof(int));
    hipMalloc(&d_val,   nnz*sizeof(float));
    hipMalloc(&d_x,     n*sizeof(float));
    hipMalloc(&d_y,     n*sizeof(float));

    hipMemcpy(d_rowptr,hA.rowptr.data(),(n+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_colind,hA.colind.data(),nnz*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_val,   hA.val.data()   ,nnz*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_x,     hx.data()       ,n*sizeof(float)  ,hipMemcpyHostToDevice);

    dim3 block(32,8);                       // 32 lanes × 8 rows
    dim3 grid ((n+block.y-1)/block.y);

    hipEvent_t t0,t1;  hipEventCreate(&t0); hipEventCreate(&t1);
    hipEventRecord(t0);
    spmv_csr_kernel<<<grid,block>>>(n,d_rowptr,d_colind,d_val,d_x,d_y);
    hipEventRecord(t1);  hipEventSynchronize(t1);
    float ms; hipEventElapsedTime(&ms,t0,t1);

    double gflops = 2.0*nnz/ms/1e6;
    std::cout << "cuda_spmv  n="<<n<<"  GFLOPs="<<gflops<<"\n";

    hipMemcpy(hy.data(),d_y,n*sizeof(float),hipMemcpyDeviceToHost);
    hipFree(d_rowptr); hipFree(d_colind); hipFree(d_val);
    hipFree(d_x); hipFree(d_y);
}